#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "test_utils.h"

// Number of inner loop iterations in the kernel (reduced from 50 to 5)
// #define INNER_LOOPS 5
#define INNER_LOOPS 50
// Total number of kernel launches (reduced from 500 to 100)
#define NUM_LAUNCHES 100

__global__ void computeKernel(double* data, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        double val = data[tid];
        for(int i = 0; i < INNER_LOOPS; ++i) { 
            val = sin(val) * cos(val);
        }
        data[tid] = val;
    }
}

int main() {
    int N = 1 << 26;
    double* d_data;

    hipError_t err = hipMalloc(&d_data, N * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory: %s\n", hipGetErrorString(err));
        return -1;
    }

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("Starting performance benchmark, executing %d kernel launches...\n", NUM_LAUNCHES);

    hipEventRecord(start);

    for (int i = 0; i < NUM_LAUNCHES; ++i) {
        computeKernel<<<blocks, threadsPerBlock>>>(d_data, N);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    float iterations_per_second = NUM_LAUNCHES / seconds;

    const char* sm_limit_env = getenv("CUDA_DEVICE_SM_LIMIT");
    printf("\n--- Benchmark Results ---\n");
    printf("SM Limit Setting:   %s%%\n", sm_limit_env ? sm_limit_env : "100 (Default)");
    printf("Total Launches:     %d\n", NUM_LAUNCHES);
    printf("Total Time (sec):   %.4f\n", seconds);
    printf("Performance:        %.2f launches/sec\n", iterations_per_second);
    printf("-------------------------\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_data);

    return 0;
}