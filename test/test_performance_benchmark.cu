#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "test_utils.h"

// Kernel 內部計算迴圈次數 (從 50 降低到 5)
#define INNER_LOOPS 5
// Kernel 總啟動次數 (從 500 降低到 200)
#define NUM_LAUNCHES 200

__global__ void computeKernel(double* data, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        double val = data[tid];
        for(int i = 0; i < INNER_LOOPS; ++i) { 
            val = sin(val) * cos(val);
        }
        data[tid] = val;
    }
}

int main() {
    int N = 1 << 26;
    double* d_data;

    hipError_t err = hipMalloc(&d_data, N * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory: %s\n", hipGetErrorString(err));
        return -1;
    }

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("開始快速效能評測，將執行 %d 次 Kernel 啟動...\n", NUM_LAUNCHES);

    hipEventRecord(start);

    for (int i = 0; i < NUM_LAUNCHES; ++i) {
        computeKernel<<<blocks, threadsPerBlock>>>(d_data, N);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    float iterations_per_second = NUM_LAUNCHES / seconds;

    const char* sm_limit_env = getenv("CUDA_DEVICE_SM_LIMIT");
    printf("\n--- 評測結果 ---\n");
    printf("算力限制設定:   %s%%\n", sm_limit_env ? sm_limit_env : "100 (Default)");
    printf("總共執行次數:   %d\n", NUM_LAUNCHES);
    printf("總耗時 (秒):    %.4f\n", seconds);
    printf("效能:           %.2f launches/sec\n", iterations_per_second);
    printf("--------------------\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_data);

    return 0;
}